#include "hip/hip_runtime.h"
// CUDA libraries.
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdio>

// Include associated header file.
#include "../include/cuda_kernel.cuh"



// TODO: Define the kernel function right here
__global__ void short_path_update_naive(short* graph, short* dist, int* u, int V){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid<V){
        dist[tid] = (short)min((dist[*u] + graph[*u*V+tid])+(short)(graph[*u*V+tid] == 0)*dist[tid], dist[tid]);
        //printf("dist[%d]= %hu\r\n",tid ,dist[tid]);
    }
}

__global__ void minDistance_kernel_naive(short* dist, bool* sptSet, int n, int V, short* out, int* out_idx,int* node_u){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int i = 2 * tid;

    if (i >= n) return;
    
    short val1 = (i < n && (n != V ||!sptSet[i])) ? dist[i] : (short)0x7FFF;
    short val2 = (i + 1 < n && ( n != V || !sptSet[i + 1])) ? dist[i + 1] : (short)0x7FFF;
    int idx1, idx2;
    if(n == V){
        idx1 = i;
        idx2 =  i + 1;
    }
    else{
        idx1 = out_idx[i];
        idx2 =  out_idx[i + 1];
    }
    

    if (val1 <= val2) {
        //printf("min(%hu,%hu) = %hu\r\n", val1, val2, val1);
        out[tid] = val1;
        out_idx[tid] = idx1;
        if(n==2){
            sptSet[idx1] = true;
            *node_u=idx1;
        }
    } else {
        //printf("min(%hu,%hu) = %hu\r\n", val1, val2, val2);
        out[tid] = val2;
        out_idx[tid] = idx2;
        if(n==2){
            /*find the minimum and update the vector*/
            sptSet[idx2] = true;
            *node_u=idx2;
        }
    }
    
    
}

int minDistance(short dist[], bool sptSet[], int V)
{
    // Initialize min value
    int min = 0x7FFF, min_index;

    for (int v = 0; v < V; v++)
        if (sptSet[v] == false && dist[v] <= min)
            min = dist[v], min_index = v;

    return min_index;
}


void dijkstra_parallelize_naive(short* graph, int src, short* dist, int V)
{   
    short *graph_d, *dist_d;
    int thread = 0;
        if(V<256)
            thread = V;
        else
            thread =256;

    hipMalloc((void **) &graph_d, V*V*sizeof(short));
    hipMalloc((void **) &dist_d, V*sizeof(short));
    hipMemcpy(graph_d, graph, V*V*sizeof(short), hipMemcpyHostToDevice);

    /*var for min*/
    int n = V;
    short* in_buff;
    bool *sptSet_d;
    short* out;
    int* outIdx;
    short *tmp_out;
    int* node_u;
    
    hipMalloc(&sptSet_d, V*sizeof(bool));
    hipMalloc(&out, (n/2+1)*sizeof(short));
    //temporary buffer to be switched with in buffer in order to get the min in divide and conquer mode
    hipMalloc((void **) &tmp_out, (n/2+1)*sizeof(short));
    hipMalloc(&outIdx, (n/2+1)*sizeof(int));
    hipMalloc(&node_u, sizeof(int));
    

    bool sptSet[V]; //Set to true all the vertexes for which the shortest path has been already found

    // Initialize all distances as INFINITE and stpSet[] as false
    for (int i = 0; i < V; i++)
        dist[i] = 0x7FFF, sptSet[i] = false;

    // Distance of source vertex from itself is always 0
    dist[src] = 0;

    hipMemcpy(dist_d, dist, V*sizeof(short), hipMemcpyHostToDevice);
    hipMemcpy(sptSet_d, sptSet, V*sizeof(bool), hipMemcpyHostToDevice);
    // Find shortest path for all vertices
    for (int count = 0; count < V - 1; count++) {
        //Pick the minimum distance vertex
        n = V;
        in_buff = dist_d;
        out;

        while (n > 1) {
            //printf("============\r\n");
            dim3 blockSize_min(thread,1,1);
            dim3 gridSize_min((n + thread*2 - 1) / (thread*2),1);
            minDistance_kernel_naive<<<gridSize_min, blockSize_min>>>(in_buff, sptSet_d, n, V, out, outIdx, node_u);
            hipDeviceSynchronize();
            // swap in/out
            in_buff = out;
            out = tmp_out;
            n = (n + 1) / 2;
        }

        //int u;
        //hipMemcpy(&u, outIdx, sizeof(int), hipMemcpyDeviceToHost);
        //int u = minDistance(dist, sptSet, V);
        //sptSet[u] = true;
        //printf(" u :%d -", u);

        
        dim3 blockSize(thread,1,1);
        dim3 gridSize(V/thread+1,1);

        short_path_update_naive<<<gridSize, blockSize>>>(graph_d, dist_d, node_u, V);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA Error: %s\n", hipGetErrorString(err));
        }
        hipDeviceSynchronize();
        /*hipMemcpy(dist, dist_d, V*sizeof(short), hipMemcpyDeviceToHost);
        printf("dist: ");
        for (int v = 0; v < V; v++)
            printf("%hu, ",dist[v]);
        printf("\r\n");*/
        
    }
    hipMemcpy(dist, dist_d, V*sizeof(short), hipMemcpyDeviceToHost);
}










